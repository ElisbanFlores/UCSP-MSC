#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <stdio.h> 
#include <type_traits> 
#include <cmath> 
#include <time.h>
#include <fstream>

#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <omp.h>


typedef double T_DOUBLE;
typedef char T_CHAR;
typedef long T_LONG;
typedef float T_FLOAT;
typedef int T_INT;
typedef unsigned char T_BYTE;

const T_LONG BLOQUELINEA = 1024;

using namespace cv;
using namespace std;

#define norm(x, y) (fabs(x) + fabs(y)) 

//Variables globales

clock_t h_tIni, h_tFin, h_tTotal; //  Para calculo de tiempo en CPU
hipEvent_t d_tIni, d_tFin; float d_tTotal; // Para calculo de tiempo en GPU


/*********************************************
* PARA VERIFICAR ERRORES DE CUDA QUE SE DESENCADENA DESDE EL HOST
*********************************************/

#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
			file, line, (int)err, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}



/**********************************************
* FUNCION PARA OBTENER EL TIEMPO EN CPU
**********************************************/

double getMilisegundos(clock_t c)
{
	double tiempo = 0;
	tiempo = ((c / (double)CLOCKS_PER_SEC) * 1000);
	return tiempo;
}


/*************************************************************
* PARTE HOST
*************************************************************/

template<class T>
class h_Matriz
{

public:
	T *ptr;
	size_t row, col;
	h_Matriz(){}
	h_Matriz(size_t n)				{	inicializar(n,n);	}
	h_Matriz(size_t m, size_t n)	{	inicializar(m, n);	}
	h_Matriz(cv::Mat img)
	{
		inicializar(img.rows, img.cols);		
		for (int i = 0; i < row; i++)		
			memcpy(&(ptr[i*col]), img.ptr<T>(i, 0), col * sizeof(T));			
	}
	
	void inicializar(size_t m, size_t n)
	{
		row = m;
		col = n;
		ptr = new T[row * col];	
	}

	inline T Get(size_t r, size_t c)			{	return *(ptr + r*(col)+c);	}	
	inline void Set(size_t r, size_t c, T val)	{	*(ptr + r*(col)+c) = val;	}
	
	void Set_Matriz(h_Matriz<T> mat) 
	{
		delete ptr;
		inicializar(mat.row, mat.col);
		memcpy(&(ptr[0]), &(mat.ptr[0]), row*col * sizeof(T));
	}
	
	void Get_Matriz(h_Matriz<T> *mat)
	{
		if (mat->row == row && mat->col == col)			
			memcpy(&mat->ptr[0], &(ptr[0]), row*col * sizeof(T));
	}
	
	void h_Matriz2Mat(cv::Mat *img)
	{
		if (img->rows == row && img->cols == col)
			for (size_t i = 0; i < row; i++)
				memcpy(img->ptr<T>(i, 0), &(ptr[i*col]), col * sizeof(T));
	}
	
	void Imprimir()
	{
		for (size_t i = 0; i < row; i++)
		{
			for (size_t j = 0; j < col; j++)
				cout << ptr[i*col + j] << "\t";
			cout << endl;
		}
	}

	~h_Matriz()
	{
	}
};

// modificar el kernel para convolucion
template<class T>
void convolucion(h_Matriz<T> *kernel)
{	 	
	int r = kernel->row ;
	int c = kernel->col;
	h_Matriz<T> temp(r,c);	
	for (int k = 0; k < r; k++)
		for (int l = 0; l < c; l++)
			temp.Set(k, l, kernel->Get(r - k-1, c - l-1));
	kernel->Set_Matriz(temp);
}

// correlacion, es convolucion si el kernel es modificado para convolucion
template<class T>
void correlacion(h_Matriz<T> *img, h_Matriz<T> *imgout, h_Matriz<T> kernel)
{		
	T suma1;
	int mitad = kernel.row / 2;	
	for (int i = mitad; i < img->row-mitad;i++)
		for (int j = mitad; j < img->col - mitad; j++)
		{
			suma1 = 0;			
			for (int k = 0; k < kernel.row; k++)
				for (int l = 0; l < kernel.col; l++)					
					suma1 += img->Get(i - mitad + k, j - mitad + l)*kernel.Get(k, l);
			imgout->Set(i, j, (T)(suma1));
		}
}

// correlacion2 con dos kernel, es convolucion si los kernel es modificado para convolucion
template<class T, class T1>
void correlacion2(h_Matriz<T> *img, h_Matriz<T> *imgout, h_Matriz<T1> kernel, h_Matriz<T1> kernel2)
{
	T suma1;
	T suma2;
	T tmp;
	int mitad = kernel.row / 2;

	for (int i = mitad; i < img->row - mitad; i++)
		for (int j = mitad; j < img->col - mitad; j++){
			suma1 = 0;
			suma2 = 0;
			for (int k = 0; k < kernel.row; k++)
				for (int l = 0; l < kernel.col; l++){
					tmp = img->Get(i - mitad+k, j - mitad+l);	
					suma1 += tmp*kernel.Get(k, l);	
					suma2 += tmp*kernel2.Get(k, l);
				}
			T val = norm((T)suma1, (T)suma2);
			imgout->Set(i, j, (T)val);
		}
}

/*************************************************************
* PARTE HOST - paralelo con OpenMP
*************************************************************/
// correlacion, es convolucion si el kernel es modificado para convolucion
template<class T>
void p_correlacion(h_Matriz<T> *img, h_Matriz<T> *imgout, h_Matriz<T> kernel)
{
	T suma1;
	int mitad = kernel.row / 2;
	int i, j, k, l;
	int ir = img->row - mitad;
	int ic = img->col - mitad;
	int kr = kernel.row;
	int kc = kernel.row;
	//kernel.Imprimir();
#pragma omp parallel  for shared(kernel, img,imgout, ir, ic,kr,kc, mitad) private( i, j, k, l, suma1) 	
	for (int i = mitad; i < ir; i++)
		for (int j = mitad; j < ic; j++)
		{
			suma1 = 0;
			for (int k = 0; k < kr; k++)
				for (int l = 0; l < kc; l++)
					suma1 += img->Get(i - mitad + k, j - mitad + l)*kernel.Get(k, l);
			//std::cout << suma1<<std::endl;
			imgout->Set(i, j, (T)(suma1));
		}
}

// correlacion2 con dos kernel, es convolucion si los kernel es modificado para convolucion
template<class T, class T1>
void p_correlacion2(h_Matriz<T> *img, h_Matriz<T> *imgout, h_Matriz<T1> kernel, h_Matriz<T1> kernel2)
{
	T suma1;
	T suma2;
	T tmp;
	T_INT mitad = kernel.row / 2;
	T val;	
	T_INT ir = img->row - mitad;
	T_INT ic = img->col - mitad;
	T_INT kr = kernel.row;
	T_INT kc = kernel.row;
	T_INT i, j, k, l;
	omp_set_num_threads(8);
#pragma omp parallel  for shared(kernel, kernel2, img,imgout, ir, ic,kr,kc, mitad) private( i, j, k, l, suma1, suma2,tmp) 
	for (i = mitad; i < ir; i++){
		for (j = mitad; j < ic; j++){
			suma1 = 0;
			suma2 = 0;
			for (k = 0; k < kr; k++){
				for (l = 0; l < kc; l++){
					tmp = img->Get(i - mitad + k, j - mitad + l);
					suma1 += tmp*kernel.Get(k, l);
					suma2 += tmp*kernel2.Get(k, l);
				}
			}
			imgout->Set(i, j, (T)norm((T)suma1, (T)suma2));
		}
	}
}
/*****************************************************************
* PARTE DEVICE
*****************************************************************/

template<class T>
class d_Matriz
{

public:
	T *d_ptr;
	size_t row, col;
	d_Matriz(h_Matriz<T> mat)
	{
		inicializar(mat);
	}

	void inicializar(h_Matriz<T> mat)
	{
		row = mat.row;
		col = mat.col;
		checkCudaErrors(hipMalloc((void**)&d_ptr, row*col * sizeof(T)));
		checkCudaErrors(hipMemcpy(d_ptr, mat.ptr, col* row*sizeof(T), hipMemcpyHostToDevice));		
	}

	__device__ inline T Get(size_t r, size_t c)				{ return *(d_ptr + r*(col)+c); }
	__device__ inline void Set(size_t r, size_t c, T val)	{ *(d_ptr + r*(col)+c) = val; }

	void Set_Matriz(h_Matriz<T> mat)
	{		
		checkCudaErrors(hipFree(d_ptr));
		inicializar(mat);		
	}
	
	void Get_Matriz(h_Matriz<T> mat)
	{
		if (mat.row == row && mat.col == col)
			checkCudaErrors(hipMemcpy(mat.ptr, d_ptr, col* row*sizeof(T), hipMemcpyDeviceToHost));
	}	

	~d_Matriz()
	{
	}
};

// modificar el kernel para convolucion
template<class T>
__device__ void d_conv3(d_Matriz<T> *kernel)
{
	int r = kernel->row - 1;
	int c = kernel->col - 1;

	d_Matriz<T> temp(3);
	for (int k = 0; k <= r; k++)
		for (int l = 0; l <= c; l++)	
			temp.Set(k, l, kernel->Get(r-k, c-l));
	kernel->Set_Matriz(temp);
}

// correlacion, es convolucion si el kernel es modificado para convolucion
template<class T>
__global__ void d_correlacion(d_Matriz<T> img, d_Matriz<T> imgout, d_Matriz<T> kernel)
{
	T suma1;	
	
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;	
	T_INT mitad = kernel.row / 2;
	while (j<(img.row - mitad)*img.col)
	{				
		suma1 = 0;
		for (T_INT k = 0; k < kernel.row; k++)
			for (T_INT l = 0; l < kernel.col; l++)
				suma1 += img.Get(i - mitad + k, j - mitad + l)*kernel.Get(k, l);
		imgout.Set(i, j, (T)(suma1));
		
		j += blockDim.x*gridDim.x;
	}
}
//__device__ void d_correlacion(d_Matriz<T> &img, d_Matriz<T> &imgout, d_Matriz<T> kernel)



// correlacion2 con dos kernel, es convolucion si los kernel es modificado para convolucion
template<class T, class T1>
__global__ void d_correlacion2(d_Matriz<T> img, d_Matriz<T> imgout, d_Matriz<T1> kernel, d_Matriz<T1> kernel2)
{
	T suma1;
	T suma2;
	T tmp;
	T_INT mitad = kernel.row / 2;
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	
	while (j<(img.row - mitad)*img.col)
	{
		suma1 = 0;
		suma2 = 0;
		for (T_INT k = 0; k < kernel.row; k++)
			for (T_INT l = 0; l < kernel.col; l++){
				tmp = img.Get(i - mitad+k, j-mitad+l);	
				suma1 += tmp*kernel.Get(k, l);	
				suma2 += tmp*kernel2.Get(k, l);
			}
			
		imgout.Set(i, j, (T)norm((T)suma1, (T)suma2));
		j += blockDim.x*gridDim.x;
	}
}

/*****************************************************************
* OBTENER FILTRO DE ARCHIVO.KER
*****************************************************************/
template<class T>
void AbrirKernel(T_CHAR *FileOrigen, h_Matriz<T> *kernel, h_Matriz<T> *kernel2, T_INT &nroKernel)
{
	std::ifstream origen(FileOrigen);

	if (origen.fail())
		std::cerr << "Error al abrir el kernel: " << FileOrigen << std::endl;
	else
	{
		T_CHAR *bloque;
		bloque = new T_CHAR[BLOQUELINEA + 1];
		T_INT m, n;
		T_FLOAT val;
		// Leer el numero de Kernel
		origen.getline(bloque, BLOQUELINEA, '\n');
		nroKernel = atoi(bloque);		
		
		if (nroKernel == 1)// 1 solo kernel
		{
			origen.getline(bloque, BLOQUELINEA, '\n');
			m = atoi(bloque);
			origen.getline(bloque, BLOQUELINEA, '\n');
			n = atoi(bloque);
			kernel->inicializar(m, n);
			kernel2->inicializar(m, n);

			for (T_INT i = 0; i < m; i++) // llenar la matriz
				for (T_INT j = 0; j < m; j++)
				{
					origen.getline(bloque, BLOQUELINEA, '\n');
					val = atof(bloque);
					kernel->Set(i, j, val);
					kernel2->Set(i, j, val);
				}		
		}
		else // se supone que son dos filtros para una misma convolucion ejemplo sobel
		{
			// para el primer kernel
			origen.getline(bloque, BLOQUELINEA, '\n');
			m = atoi(bloque);
			origen.getline(bloque, BLOQUELINEA, '\n');
			n = atoi(bloque);
			kernel->inicializar(m, n);			

			for (T_INT i = 0; i < m; i++) // llenar la matriz
				for (T_INT j = 0; j < m; j++)
				{
					origen.getline(bloque, BLOQUELINEA, '\n');
					val = atof(bloque);
					kernel->Set(i, j, val);			
				}
			
			// para el segundo kernel
			origen.getline(bloque, BLOQUELINEA, '\n');
			m = atoi(bloque);
			origen.getline(bloque, BLOQUELINEA, '\n');
			n = atoi(bloque);
			kernel2->inicializar(m, n);

			for (T_INT i = 0; i < m; i++) // llenar la matriz
				for (T_INT j = 0; j < m; j++)
				{
					origen.getline(bloque, BLOQUELINEA, '\n');
					val = atof(bloque);
					kernel2->Set(i, j, val);
				}
		}		
	}
}



int main() // main en video
{

	//namedWindow("ventana", CV_WINDOW_AUTOSIZE);	
	//cargar el archivo de video especificado
	cv::VideoCapture cvideo("video2.mp4");	
	
	//verificar si se ha podio cargar el video
	if (!cvideo.isOpened())	return -1;
	// obtener los cuadros por segundo
	T_DOUBLE fps = cvideo.get(CV_CAP_PROP_FPS);
	T_DOUBLE nf = cvideo.get(CV_CAP_PROP_FRAME_COUNT);
	cout << "Nro de frames: " << nf<< endl;
		cout<<"Nro frames por segundos: " <<fps << endl;
	
	// calcular el tiempo de espera entre cada imagen a mostrar
	//int delay = 1000 / fps;
	T_INT delay = 1;
	h_Matriz<float> kernelx;
	h_Matriz<float> kernely;

	
	/**********************************************
	* M O D O
	**********************************************/
	int modo = 3;// CPU:1, PAR:2, GPU:3
	int filtros = 1;


	//------------------------------------------------
	// OBTENER KERNEL PARA FILTRO
	//------------------------------------------------
	
	char *modoNombre;
	modoNombre = new char[250];
	
	//strcpy(modoNombre, "sobel.ker"); // 3x3 dos filtros
	//strcpy(modoNombre, "repujado.ker");// 3x3 un filtro

	//strcpy(modoNombre, "media3.ker");
	//strcpy(modoNombre, "media5.ker");
	//strcpy(modoNombre, "media11.ker");
	//strcpy(modoNombre, "media15.ker");	
	strcpy(modoNombre, "media25.ker");

	
	AbrirKernel<T_FLOAT>(modoNombre,&kernelx,&kernely,filtros);
	
	std::cout << "Kernel: "<<modoNombre <<std::endl;
	kernelx.Imprimir();
	kernely.Imprimir();
	std::cout << "presione g para correr en gpu," << std::endl << " p para correr en cpu paralelo y " << std::endl << " c para correr en cpu secuencial...." << std::endl<<" esc para salir... o esperar que termine el video...";
	getchar();
	
	// para cambiar el kernel para obtener kernel para convolution

	convolucion(&kernelx);
	convolucion(&kernely);
	//std:: cout << "Kernel con covolucion"<<std::endl;
	//kernelx.Imprimir();
	//kernely.Imprimir();

	// Para el DEVICE
	
	d_Matriz<float> d_kernelx(kernelx);
	d_Matriz<float> d_kernely(kernely);

	//d_correlacion2<float> << <imagen.col, 1024 >> >(d_imagen, d_imagenout, d_kernelx, d_kernely);	
	//d_imagenout.Get_Matriz(imagenout);

	Mat cvimagen, cvimageng ;
	double contf = 0;

	while (contf <nf)
	{		
		cvideo >> cvimagen;	
		
		cv::cvtColor(cvimagen, cvimageng, CV_BGR2GRAY);				
		cvimageng.convertTo(cvimageng, CV_32FC1);	
		
		// en HOST		
		h_Matriz<float> imagen(cvimageng);
		h_Matriz<float> imagenout(imagen.row, imagen.col);
		//cargar el primer cuadro o imagen del video en frame
			

		switch (modo)
		{
		
			case 1:
			{
				h_tIni = clock();
				if (filtros==2)
					correlacion2(&imagen, &imagenout, kernelx, kernely);
				else
					correlacion(&imagen, &imagenout, kernelx);
				h_tFin = clock();
				strcpy(modoNombre, "Salida");
				cout << "CPU : " << getMilisegundos(h_tFin - h_tIni) << "ms por frame" << endl;
				break;
			}

			case 2:
			{
				h_tIni = clock();
				if (filtros == 2)
					p_correlacion2(&imagen, &imagenout, kernelx, kernely);
				else
					p_correlacion(&imagen, &imagenout, kernelx);
				h_tFin = clock();
				strcpy(modoNombre, "Salida");
				cout << "CPU Paralela: " << getMilisegundos(h_tFin - h_tIni) << "ms por frame" << endl;
				break;
			}

			case 3:
			{
				float d_ttemp;
				hipEventCreate(&d_tIni);
				hipEventCreate(&d_tFin);
				hipEventRecord(d_tIni, 0);
				d_Matriz<float> d_imagen(imagen);
				d_Matriz<float> d_imagenout(imagenout);
				if (filtros == 2)
					d_correlacion2<float> <<<imagen.col, imagen.row / 2 >>>(d_imagen, d_imagenout, d_kernelx, d_kernely);
				else
					d_correlacion<float> <<<imagen.col, imagen.row / 2 >>>(d_imagen, d_imagenout, d_kernelx);
				d_imagenout.Get_Matriz(imagenout);				
				checkCudaErrors(hipFree(d_imagen.d_ptr));
				checkCudaErrors(hipFree(d_imagenout.d_ptr));

				hipEventRecord(d_tFin, 0);
				hipEventSynchronize(d_tFin);
				hipEventElapsedTime(&d_ttemp, d_tIni, d_tFin);
				strcpy(modoNombre, "Salida");
				cout << "GPU: " << d_ttemp << "ms por frame" << endl;
				break;
			}
			default:
				break;

		}		
		
		imagenout.h_Matriz2Mat(&cvimageng);
		
		cvimageng.convertTo(cvimageng, CV_8UC1);
		cv::imshow(modoNombre, cvimageng);
		
		//esperar un periodo de tiempo especificado por delay 
		//si se presiona la tecla 27 (ESC) salir del loop
		uchar tec=cv::waitKey(delay);
		//cout << tec<<endl;

		if (tec == 99) modo = 1;
		if (tec == 112) modo = 2;
		if (tec == 103) modo = 3;

		if (tec == 27 ) break;
		contf++;
		delete imagen.ptr;
		delete imagenout.ptr;
	}
	cout << "Ups, se termino el video" << endl;
	cv::waitKey();
	cv::destroyWindow("ventana");

}

int mainimagen() // main en imagen
{
		// calcular el tiempo de espera entre cada imagen a mostrar
	//int delay = 1000 / fps;
	T_INT delay = 1;
	h_Matriz<float> kernelx;
	h_Matriz<float> kernely;


	/**********************************************
	* M O D O
	**********************************************/
	int modo = 3;// CPU:1, PAR:2, GPU:3
	int filtros = 1;


	//------------------------------------------------
	// OBTENER KERNEL PARA FILTRO
	//------------------------------------------------

	char *modoNombre;
	modoNombre = new char[250];

	//strcpy(modoNombre, "sobel.ker"); // 3x3 dos filtros
	//strcpy(modoNombre, "repujado.ker");// 3x3 un filtro

	//strcpy(modoNombre, "media3.ker");
	//strcpy(modoNombre, "media5.ker");
	//strcpy(modoNombre, "media11.ker");
	//strcpy(modoNombre, "media15.ker");	
	strcpy(modoNombre, "media25.ker");
	

	AbrirKernel<T_FLOAT>(modoNombre, &kernelx, &kernely, filtros);

	std::cout << "Kernel: " << modoNombre << std::endl;
	kernelx.Imprimir();
	kernely.Imprimir();

	
	// para cambiar el kernel para obtener kernel para convolution

	convolucion(&kernelx);
	convolucion(&kernely);
	//std::cout << "Kernel con covolucion" << std::endl;
	//kernelx.Imprimir();
	//kernely.Imprimir();

	

	// Para el DEVICE

	d_Matriz<float> d_kernelx(kernelx);
	d_Matriz<float> d_kernely(kernely);
		
	Mat cvimagen, cvimageng;
	double contf = 0;

	cvimagen = imread("alpaca1000.jpg", 1);
	cv::cvtColor(cvimagen, cvimageng, CV_BGR2GRAY);
	cvimageng.convertTo(cvimageng, CV_32FC1);

	// en HOST		
	h_Matriz<float> imagen(cvimageng);
	h_Matriz<float> imagenout(imagen.row, imagen.col);
	//cargar el primer cuadro o imagen del video en frame
	
	
	//---------------------------------------
	// Corrida en CPU un solo nucleo
	//---------------------------------------
	h_tIni = clock();
	if (filtros == 2)
		correlacion2(&imagen, &imagenout, kernelx, kernely);
	else
		correlacion(&imagen, &imagenout, kernelx);
	h_tFin = clock();
	strcpy(modoNombre, "Salida");
	std::cout << "CPU : " << getMilisegundos(h_tFin - h_tIni) << "ms por frame" << endl;
	
	
	//---------------------------------------
	// Corrida en CPU Paralela Utilizando OpenMP 
	//---------------------------------------
	h_tIni = clock();
	if (filtros == 2)
		p_correlacion2(&imagen, &imagenout, kernelx, kernely);
	else
		p_correlacion(&imagen, &imagenout, kernelx);
	h_tFin = clock();
	strcpy(modoNombre, "Salida");
	std::cout << "CPU Paralela: " << getMilisegundos(h_tFin - h_tIni) << "ms por frame" << endl;
	
	

	//---------------------------------------
	// Corrida en GPU , el tiempo incluye copia de archivo a memorua GPU y viceversa
	//---------------------------------------
	float d_ttemp;
	hipEventCreate(&d_tIni);
	hipEventCreate(&d_tFin);
	hipEventRecord(d_tIni, 0);
	d_Matriz<float> d_imagen(imagen);
	d_Matriz<float> d_imagenout(imagenout);
	if (filtros == 2)
		d_correlacion2<float> << <imagen.col, imagen.row / 2 >> >(d_imagen, d_imagenout, d_kernelx, d_kernely);
	else
		d_correlacion<float> << <imagen.col, imagen.row / 2 >> >(d_imagen, d_imagenout, d_kernelx);
	d_imagenout.Get_Matriz(imagenout);
	checkCudaErrors(hipFree(d_imagen.d_ptr));
	checkCudaErrors(hipFree(d_imagenout.d_ptr));

	hipEventRecord(d_tFin, 0);
	hipEventSynchronize(d_tFin);
	hipEventElapsedTime(&d_ttemp, d_tIni, d_tFin);
	strcpy(modoNombre, "Salida");
	std::cout << "GPU: " << d_ttemp << "ms por frame" << endl;
	
	
	imagenout.h_Matriz2Mat(&cvimageng);

	cvimageng.convertTo(cvimageng, CV_8UC1);
	cv::imshow(modoNombre, cvimageng);
	
	cv::waitKey();
	delete imagen.ptr;
	delete imagenout.ptr;
	std::cout << "Ups, se termino....." << endl;
	cv::destroyWindow("ventana");	
	return 0;
}



